#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

/*****************************************************************************
 *
 * 
 * 
 * 
 * 
 * Compile with:
 *   nvcc -o cudapassword 2initialpass_cuda.cu
 * 
 * Dr Kevan Buckley, University of Wolverhampton, 2018 
 ****************************************************************************/

__device__ int is_a_match (char*attempt){
	char plain_password1[] = "RA34";
	char plain_password2[] = "SR56";
	char plain_password3[] = "HV70";
	char plain_password4[] = "TI52";


char *a=attempt;
char *b=attempt;
char *c=attempt;
char *d=attempt;
char *p1=plain_password1;
char *p2=plain_password2;
char *p3=plain_password3;
char *p4=plain_password4;

	while (*a == *p1){
	if (*a == '\0')
	{
	printf("found password: %s\n",plain_password1);
	break;
	}
	a++;
	p1++;
	}

	while (*b == *p2){
	if (*b == '\0')
	{
	printf("found password: %s\n",plain_password2);
	break;
	}
	b++;
	p2++;
	}

	while (*c == *p3){
	if (*c == '\0')
	{
	printf("found password: %s\n",plain_password3);
	break;
	}
	c++;
	p3++;
	}

	while (*d == *p4){
	if (*d == '\0')
	{
	printf("found password: %s\n",plain_password4);
	break;
	}
	d++;
	p4++;
	}
	return 0;
 }



__global__ void kernel (){
char s,a;
char password[5];
password [4] = '\0';

int i = threadIdx.x+65;
int j = threadIdx.y+65;
char firstvalue = i ; 
char secondvalue = j ;
password[0] = firstvalue ;
password [1] = secondvalue;

 for (s='0';s<='9'; s++){
for (a='0';a<='9'; a++){

password[2]= s;
password[3]= a;


is_a_match(password);


 }
 }
 }




int time_difference(struct timespec *start,
 struct timespec *finish, 
  long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}

int main() {
 struct timespec start, finish;
long long int time_elapsed;
clock_gettime(CLOCK_MONOTONIC, &start);
dim3 dim (26,26);
kernel <<<1,dim>>>();
hipDeviceSynchronize();

clock_gettime(CLOCK_MONOTONIC, &finish);
time_difference (&start, &finish, &time_elapsed);
printf("Time elapsed was %lldns or %0.9fs\n", time_elapsed, (time_elapsed/1.0e9));

return 0;
}





























